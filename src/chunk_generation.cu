#include "hip/hip_runtime.h"
#include "chunk_generation.cuh"

#include <iostream>

#include "db_perlin.hpp"

#include <hip/hip_runtime.h>
#include <>

void generateChunk(Octree* octree, int x_, int y_, int z_, unsigned int gridSize, unsigned int blockSize, int offsetX, int offsetY) {

    size_t numBlocks = CHUNK_W * CHUNK_W * CHUNK_H;

    thrust::device_vector<Block> chunkBlocks(numBlocks);

    for (int x = 0; x < CHUNK_W; x++) {
        for (int y = 0; y < CHUNK_H; y++) {
            for (int z = 0; z < CHUNK_W; z++) {

                float val = db::perlin((float(x) + float(x_) * CHUNK_W) / smoothing + offsetX, (float(z) + float(z_) * CHUNK_W) / smoothing + offsetY) * amplify;

                //octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 1, octree->root);
                //return;

                if (y >= val + 30) { //y >= val + 20

                    if(y <= val + 20.5 + 10){
                        chunkBlocks[x + y * CHUNK_W + z * CHUNK_W * CHUNK_H] = Block(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 1); // TODO: change y to be relative too (cube chunks)
                    }

                    //     octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 1);
                    // else if(y <= val + 20.9 + 10)
                    //     octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 3);
                    // else if (y <= val + 22 + 10)
                    //     octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 2);
                    // else if (y <= val + 45 + 10)
                    //     octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 2);
                    //else
                    //    octree->insert(x + x_ * CHUNK_W, y, z + z_ * CHUNK_W, 5, octree->root);
                }
            }
        }
    }

    insert(octree, chunkBlocks, numBlocks, gridSize, blockSize);

    // TODO: use morton codes for hashing
    generatedChunks.insert(make_pair(make_pair(x_, z_), true));
}

void generateVisibleChunks(Octree* octree) {

    map<pair<int, int>, bool>::const_iterator got;

    got = generatedChunks.find(make_pair((int)cameraPos.x / CHUNK_W, (int)cameraPos.z / CHUNK_W));
    if (got == generatedChunks.end())
        generateChunk(octree, (int)cameraPos.x % CHUNK_W, 0, (int)cameraPos.z % CHUNK_W);

    float step = 0.5;

    float focalLengthX = FOCAL_LENGTH * cos(cameraAngle.x);

    float leftAngle = angleNormalize(cameraAngle.y + halfHorFOV);
    float rightAngle = angleNormalize(cameraAngle.y - halfHorFOV);

    //cout << sin(leftAngle) << " " << cos(leftAngle) << " " << sin(rightAngle) << " " << sin(rightAngle) << endl;

    float sX1 = sin(leftAngle) * step;
    float sZ1 = cos(leftAngle) * step;

    float sX2 = sin(rightAngle) * step;
    float sZ2 = cos(rightAngle) * step;

    float posX1 = cameraPos.x / (float)CHUNK_W, posZ1 = cameraPos.z / (float)CHUNK_W, posX2 = cameraPos.x / (float)CHUNK_W, posZ2 = cameraPos.z / (float)CHUNK_W;

    float distance = 0;

    //cout << sX1 << " " << sZ1 << " " << sX2 << " " << sZ2 << endl;

    while (distance < RENDER_DISTANCE) {

        posX1 += sX1;
        posZ1 += sZ1;

        posX2 += sX2;
        posZ2 += sZ2;

        vector<pair<int, int>> points = LinePoints((int)posX1, (int)posZ1, (int)posX2, (int)posZ2);

        /*for(int i = 0; i < points.size(); i++)
            cout << "(" << int(points[i].first) << " " << int(points[i].second) << ") ";
        cout << endl;*/

        for (int i = 0; i < points.size(); i++) {

            got = generatedChunks.find(make_pair(points[i].first, points[i].second));

            if (got == generatedChunks.end())
                generateChunk(octree, points[i].first, 0, points[i].second);
        }

        distance += step;
    }
}
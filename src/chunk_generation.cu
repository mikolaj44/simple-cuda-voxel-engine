#include "hip/hip_runtime.h"
#include "chunk_generation.cuh"

#include <iostream>

#include <hip/hip_runtime.h>
#include <>

void generateChunks(Octree* octree, Vector3 cameraPos, dim3 gridSize, dim3 blockSize){

    octree->xMin = cameraPos.x - CHUNK_W;
    octree->yMin = cameraPos.y - CHUNK_W;
    octree->zMin = cameraPos.z - CHUNK_W;
    octree->level = log2(RENDER_DISTANCE_CHUNKS * CHUNK_W * 2) - 1;

    generateChunksKernel<<<gridSize, blockSize>>>(octree, octree->nodeMap.ref(cuco::insert), cameraPos);
    hipDeviceSynchronize(); // maybe remove this later
}